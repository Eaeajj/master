#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "helpers.h"
#include "calcs.h"
#include <crt/math_functions.hpp>
#include "cuda_helpers.h"
#include <stdio.h>

void reduceWithCuda1(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reduce1 <<<blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

void reduceWithCuda2(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reduce2 <<<blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

void reduceWithCuda3(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reduce3 <<<blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

void reduceWithCuda4(dim3 blocks, dim3 threads, int threadSize, int* dev_a, int* dev_c, int size) {
	reduce4 <<<blocks, threads, threadSize * sizeof(int) >> > (dev_a, dev_c, size);
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t reduceWithCuda(int* c, const int* a, unsigned int size, int threadSize, int blockSize, void(*kernel) (dim3, dim3, int, int*, int*, int))
{
	hipDeviceProp_t	devProp;
	hipGetDeviceProperties(&devProp, 0);
	int* dev_a = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	cudaStatus = hipMalloc((void**)&dev_c, blockSize * sizeof(int));
	checkError(cudaStatus, "hipMemcpy failed!", dev_c, dev_c);

	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	// Launch a kernel on the GPU with one thread for each element.
	hipEvent_t start, stop;		//��������� ���������� ����  hipEvent_t 
	float       gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	dim3 threads(threadSize, 1, 1);
	dim3 blocks(blockSize, 1, 1);
	kernel(blocks, threads, threadSize, dev_a, dev_c, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "reductionMin4 launch failed: %s\n", dev_a, dev_c);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching reductionMin4!\n", dev_a, dev_c);

	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time: %.5f ms\n", gpuTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, blockSize * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", dev_a, dev_c);

	for (size_t i = 1; i < blockSize; i++) {
		if (c[i] < c[0]) {
			c[0] = c[i];
		}
	}
	return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogramWithCuda(int* c, const int* a, unsigned int size, int threadSize)
{
	int NUM_BINS = 256;
	int PART_SIZE = size < 300000 ? 3000 : 10000;
	int NUM_PARTS = ceil(double(size) / PART_SIZE);
	int* input = 0;
	int* dev_c = 0;
	int* result = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkError(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", input, dev_c);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, 256 * NUM_PARTS * sizeof(int));
	checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

	cudaStatus = hipMalloc((void**)&result, 256 * sizeof(int));
	checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

	cudaStatus = hipMalloc((void**)&input, size * sizeof(int));
	checkError(cudaStatus, "hipMalloc failed!", input, dev_c);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(input, a, size * sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus, "hipMemcpy failed!", input, dev_c);

	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	histogram<< <NUM_PARTS, threadSize, NUM_BINS * sizeof(int) >> > (input, size, NUM_BINS, dev_c);
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "histogramSmemAtomics launch failed: %s\n", input, dev_c);

	histogramMerge<< <2, NUM_BINS >> > (dev_c, NUM_BINS, NUM_PARTS, result);
	cudaStatus = hipGetLastError();
	checkError(cudaStatus, "histogramFinalAccum launch failed: %s\n", input, dev_c);

	cudaStatus = hipDeviceSynchronize();
	checkError(cudaStatus, "hipDeviceSynchronize returned error code %d after launching multiplyKernel!\n", input, dev_c);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time: %.5f ms\n", gpuTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, result, 256 * sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus, "hipMemcpy failed!", input, dev_c);

	return cudaStatus;
}